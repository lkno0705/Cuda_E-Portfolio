
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <stdlib.h>

const int m = 14400;
const int n = 14400;

// CUDA Kernel for MatrixAddition
__global__ void MatAdd(float* A, float* B, float* C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculates Current Index
    if (i < m*n)
        C[i] = A[i] + B[i];
}

float* createRandomMatrix(float *matrix, int m, int n) {
    matrix = new float[m * n];
    for (int r = 0; r < m; r++) {
        for (int c = 0; c < n; c++) {
            matrix[n * r + c] = static_cast <float> (rand() % 10) / 1.0;
        }
    }
    return matrix;
}

float* createEmptyMatrix(float* matrix, int m, int n) {
    matrix = new float[m * n];
    for (int r = 0; r < m; r++) {
        for (int c = 0; c < n; c++) {
            matrix[n * r + c] = 0.0;
        }
    }
    return matrix;
}

int main() {
    float* A = (float*)malloc(m * n);
    float* B = (float*)malloc(m * n);
    float* C = (float*)malloc(m * n);

    float* d_A;
    float* d_B;
    float* d_C;

    auto start1 = std::chrono::high_resolution_clock::now();
    std::cout << "[+] Generation of Matrices started \n";
    A = createRandomMatrix(A, m, n);
    std::cout << "[+] Generation of Matrix A finished \n";
    B = createRandomMatrix(B, m, n);
    std::cout << "[+] Generation of Matrix B finished \n";
    C = createEmptyMatrix(C, m, n);
    std::cout << "[+] Generation of Matrix C finished \n";
    auto stop1 = std::chrono::high_resolution_clock::now();

    std::cout << "[+] Generation on CPU finished \n[+] Duration: " << std::chrono::duration<double>(stop1 - start1).count() << " seconds\n";

    int blockSize = 64; // Block Size of GPU, 64 for RTX 2070super
    int numBlocks = ((n*m) + blockSize - 1) / blockSize; // Calculates the number of Blocks

    // Allocate Memory on GPU
    hipMalloc(&d_A, (m * n) * sizeof(float));
    hipMalloc(&d_B, (m * n) * sizeof(float));
    hipMalloc(&d_C, (m * n) * sizeof(float));

    // Copy Data to GPU
    hipMemcpy(d_A, A, (m * n) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, (m * n) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, (m * n) * sizeof(float), hipMemcpyHostToDevice);

    std::cout << "[+] Using " << numBlocks << " Blocks with " << blockSize << " Threads\n";
    std::cout << "[+] Calculation started with " << (numBlocks * blockSize) << " Threads";
    auto start = std::chrono::high_resolution_clock::now();

    // Start Kernel
    MatAdd<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

    // Wait for Calculation to finish
    hipDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();

    // Copy result to Host
    hipMemcpy(C, d_C, (m * n) * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "\n[+] Multithreaded calculation finished \n[+] Duration: " << std::chrono::duration<double>(stop - start).count() << " seconds";

    // Free Memory on GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free memory on host
    delete[] A;
    delete[] B;
    delete[] C;
}
